#include <stdio.h>

#include "hip/hip_runtime.h"


__global__ void PrintThreadIDs() {
  printf("thread.x: %d, thread.y: %d, thread.z: %d\n", threadIdx.x, threadIdx.y, threadIdx.z);
}

int main(int argc, const char* argv[]) {
  int nx = 16;
  int ny = 16;

  dim3 block(8, 8);
  dim3 grid(nx / 8, ny / 8);
  PrintThreadIDs<<<grid, block>>>();

  hipDeviceSynchronize();
  hipDeviceReset();
  return 0;
}