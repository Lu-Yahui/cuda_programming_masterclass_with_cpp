#include <stdio.h>

#include "hip/hip_runtime.h"


__global__ void PrintDetails() {
  printf(
      "blockIdx.x %d, blockIdx.y %d, blockIdx.z %d, blockDim.x %d blockDim.y %d, blockDim.z %d, gridDim.x %d gridDim.y "
      "%d, gridDim.y %d\n",
      blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z);
}

int main(int argc, const char* argv[]) {
  int nx = 16;
  int ny = 16;
  dim3 block(8, 8);
  dim3 grid(nx / 8, ny / 8);
  PrintDetails<<<grid, block>>>();
  hipDeviceSynchronize();
  hipDeviceReset();
  return 0;
}