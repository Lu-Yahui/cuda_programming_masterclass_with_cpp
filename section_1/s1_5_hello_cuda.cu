#include <stdio.h>

#include "hip/hip_runtime.h"


__global__ void HelloCuda() {
  printf("Hello CUDA world\n");
}

int main(int argc, const char* argv[]) {
  HelloCuda<<<1, 1>>>();
  hipDeviceSynchronize();
  hipDeviceReset();
  return 0;
}
