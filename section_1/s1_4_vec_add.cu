#include <stdio.h>

#include "hip/hip_runtime.h"


__global__ void AddKernel(int* c, const int* a, const int* b) {
  int i = threadIdx.x;
  c[i] = a[i] + b[i];
}

hipError_t AddWithCuda(int* c, const int* a, const int* b, unsigned int size) {
  int* d_a;
  int* d_b;
  int* d_c;

  hipError_t cuda_status;

  // Choose which GPU to run on, change this on a multi-GPU system.
  cuda_status = hipSetDevice(0);
  if (cuda_status != hipSuccess) {
    fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    goto Error;
  }

  // Allocate GPU buffers for three vectors (two input, one output).
  cuda_status = hipMalloc((void**)&d_c, size * sizeof(int));
  if (cuda_status != hipSuccess) {
    fprintf(stderr, "hipMalloc failed!");
    goto Error;
  }

  cuda_status = hipMalloc((void**)&d_a, size * sizeof(int));
  if (cuda_status != hipSuccess) {
    fprintf(stderr, "hipMalloc failed!");
    goto Error;
  }

  cuda_status = hipMalloc((void**)&d_b, size * sizeof(int));
  if (cuda_status != hipSuccess) {
    fprintf(stderr, "hipMalloc failed!");
    goto Error;
  }

  // Copy input vectors from host memory to GPU buffers.
  cuda_status = hipMemcpy(d_a, a, size * sizeof(int), hipMemcpyHostToDevice);
  if (cuda_status != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed!");
    goto Error;
  }

  cuda_status = hipMemcpy(d_b, b, size * sizeof(int), hipMemcpyHostToDevice);
  if (cuda_status != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed!");
    goto Error;
  }

  // Launch a kernel on the GPU with one thread for each element.
  AddKernel<<<1, size>>>(d_c, d_a, d_b);
  cuda_status = hipGetLastError();
  if (cuda_status != hipSuccess) {
    fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cuda_status));
    goto Error;
  }

  // cudaDeviceSynchronize waits for the kernel to finish, and returns
  // any errors encountered during the launch.
  cuda_status = hipDeviceSynchronize();
  if (cuda_status != hipSuccess) {
    fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cuda_status);
    goto Error;
  }

  // Copy output vector from GPU buffer to host memory.
  cuda_status = hipMemcpy(c, d_c, size * sizeof(int), hipMemcpyDeviceToHost);
  if (cuda_status != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed!");
    goto Error;
  }

Error:
  hipFree(d_c);
  hipFree(d_a);
  hipFree(d_b);

  return cuda_status;
}

int main(int argc, const char* argv[]) {
  const int array_size = 5;
  const int a[array_size] = {1, 2, 3, 4, 5};
  const int b[array_size] = {10, 20, 30, 40, 50};
  int c[array_size];

  hipError_t cuda_status = AddWithCuda(c, a, b, array_size);
  if (cuda_status != hipSuccess) {
    fprintf(stderr, "addWithCuda failed!");
    return 1;
  }

  printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n", c[0], c[1], c[2], c[3], c[4]);

  // cudaDeviceReset must be called before exiting in order for profiling and
  // tracing tools such as Nsight and Visual Profiler to show complete traces.
  cuda_status = hipDeviceReset();
  if (cuda_status != hipSuccess) {
    fprintf(stderr, "hipDeviceReset failed!");
    return 1;
  }

  return 0;
}