#include "hip/hip_runtime.h"
#include <cmath>

#include "vec_add_example.h"

__global__ void VecAddKernel(float* A, float* B, float* C, int n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    C[i] = A[i] + B[i];
  }
}

void VecAddGpu(float* h_A, float* h_B, float* h_C, int n) {
  int size = n * sizeof(float);

  float* d_A;
  float* d_B;
  float* d_C;
  hipMalloc((void**)&d_A, size);
  hipMalloc((void**)&d_B, size);
  hipMalloc((void**)&d_C, size);

  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  int block_size = static_cast<int>(std::ceil(n / 256.0));
  dim3 dim_grid(block_size, 1, 1);
  dim3 dim_block(256, 1, 1);
  VecAddKernel<<<dim_grid, dim_block>>>(d_A, d_B, d_C, n);

  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}
